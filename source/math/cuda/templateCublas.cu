#include "hip/hip_runtime.h"
/** \file templateCublas.cu
  * \author Jared R. Males
  * \brief Implementation of the template interface to cuBlas
  * \ingroup cuda_files
  *
  */

//***********************************************************************//
// Copyright 2020 Jared R. Males (jaredmales@gmail.com)
//
// This file is part of mxlib.
//
// mxlib is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// mxlib is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with mxlib.  If not, see <http://www.gnu.org/licenses/>.
//***********************************************************************//

#include "math/cuda/templateCublas.hpp"
#include <iostream>

namespace mx
{
namespace cuda
{

//----------------------------------------------------
// Tscal

template<>
hipblasStatus_t cublasTscal<float>( hipblasHandle_t handle, 
                                   int n,
                                   const float *alpha,
                                   float *x, 
                                   int incx
                                 )
{
   return ::hipblasSscal( handle, n, alpha, x, incx);
}

template<>
hipblasStatus_t cublasTscal<double>( hipblasHandle_t handle, 
                                    int n,
                                    const double *alpha,
                                    double *x, 
                                    int incx
                                  )
{
   return ::hipblasDscal( handle, n, alpha, x, incx);
}

template<>
hipblasStatus_t cublasTscal<hipComplex>( hipblasHandle_t handle, 
                                       int n,
                                       const hipComplex *alpha,
                                       hipComplex *x, 
                                       int incx
                                     )
{
   return ::hipblasCscal( handle, n, alpha, x, incx);
}

template<>
hipblasStatus_t cublasTscal<hipDoubleComplex>( hipblasHandle_t handle, 
                                             int n,
                                             const hipDoubleComplex *alpha,
                                             hipDoubleComplex *x, 
                                             int incx
                                           )
{
   return ::hipblasZscal( handle, n, alpha, x, incx);
}

//----------------------------------------------------
// Taxpy

template<>
hipblasStatus_t cublasTaxpy<float>( hipblasHandle_t handle, 
                                   int n,
                                   const float *alpha,
                                   const float *x, 
                                   int incx,
                                   float *y, 
                                   int incy
                                 )
{
   return ::hipblasSaxpy(handle, n, alpha, x, incx, y, incy);
}

template<>
hipblasStatus_t cublasTaxpy<double>( hipblasHandle_t handle, 
                                    int n,
                                    const double *alpha,
                                    const double *x, 
                                    int incx,
                                    double *y, 
                                    int incy
                                  )
{
   return ::hipblasDaxpy(handle, n, alpha, x, incx, y, incy);
}
 
template<>
hipblasStatus_t cublasTaxpy<hipComplex>( hipblasHandle_t handle, 
                                       int n,
                                       const hipComplex *alpha,
                                       const hipComplex *x, 
                                       int incx,
                                       hipComplex *y, 
                                       int incy
                                     )
{
   return ::hipblasCaxpy(handle, n, alpha, x, incx, y, incy);
}

template<>
hipblasStatus_t cublasTaxpy<hipDoubleComplex>( hipblasHandle_t handle, 
                                             int n,
                                             const hipDoubleComplex *alpha,
                                             const hipDoubleComplex *x, 
                                             int incx,
                                             hipDoubleComplex *y, 
                                             int incy
                                           )
{
   return ::hipblasZaxpy(handle, n, alpha, x, incx, y, incy);
}

//----------------------------------------------------
// Element-wise (Hadamard) products of vectors

   
// \test Scenario: multiplying two vector element by element \ref test_math_templateCublas_elementwiseXxY "[test doc]"
template<typename dataT1, typename dataT2>
__device__
dataT1 elementMul( dataT1 & a, 
                   dataT2 & b
                 )
{
    return a*b;
}

// complex-float by complex-float multiplication
// \test Scenario: multiplying two vector element by element \ref test_math_templateCublas_elementwiseXxY "[test doc]"
template<>
__device__ 
hipComplex elementMul<hipComplex, hipComplex>( hipComplex & a, 
                                            hipComplex & b
                                          )
{
    hipComplex c;
    
    ((float*) &c)[0] = ((float*) &a)[0] * ((float*) &b)[0] - ((float*) &a)[1] * ((float*) &b)[1];
    ((float*) &c)[1] = ((float*) &a)[0] * ((float*) &b)[1] + ((float*) &a)[1] * ((float*) &b)[0];
    return c;

    
}

// complex-float by scalar multiplication
// \test Scenario: multiplying two vector element by element \ref test_math_templateCublas_elementwiseXxY "[test doc]"
template<>
__device__
hipComplex elementMul<hipComplex, float>( hipComplex & a, 
                                        float & b
                                      )
{
    hipComplex c;
    
    ((float*) &c)[0] = ((float*) &a)[0] * b; 
    ((float*) &c)[1] = ((float*) &a)[1] * b; 
    return c;

    
}


// complex-double by complex-double multiplication
// \test Scenario: multiplying two vector element by element \ref test_math_templateCublas_elementwiseXxY "[test doc]"
template<>
__device__ 
hipDoubleComplex elementMul<hipDoubleComplex, hipDoubleComplex>( hipDoubleComplex & a, 
                                                              hipDoubleComplex & b
                                                            )
{
    hipDoubleComplex c;
    
    ((double*) &c)[0] = ((double*) &a)[0] * ((double*) &b)[0] - ((double*) &a)[1] * ((double*) &b)[1];
    ((double*) &c)[1] = ((double*) &a)[0] * ((double*) &b)[1] + ((double*) &a)[1] * ((double*) &b)[0];
    return c;

    
}

// complex-double by real-double multiplication
// \test Scenario: multiplying two vector element by element \ref test_math_templateCublas_elementwiseXxY "[test doc]"
template<>
__device__
hipDoubleComplex elementMul<hipDoubleComplex, double>( hipDoubleComplex & a, 
                                                     double & b
                                                   )
{
    hipDoubleComplex c;
    
    ((double*) &c)[0] = ((double*) &a)[0] * b; 
    ((double*) &c)[1] = ((double*) &a)[1] * b;
    
    return c;

    
}

// \test Scenario: multiplying two vector element by element \ref test_math_templateCublas_elementwiseXxY "[test doc]"
template<typename dataT1, typename dataT2>
__global__ 
void elwiseMul(dataT1 *a, dataT2 *b, int size)
{   
   #ifdef __HIPCC__

   const int numThreads = blockDim.x * gridDim.x;
   const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

   for (int i = threadID; i < size; i += numThreads)
   {
       a[i] = elementMul<dataT1, dataT2>( a[i],  b[i]);
   }
    
   #endif //__HIPCC__
}

// Calculates the element-wise product of two vectors, storing the result in the first.
/* Calculates x = x * y element by element, a.k.a. the Hadamard product.
 * \test Scenario: multiplying two vector element by element \ref test_math_templateCublas_elementwiseXxY "[test doc]"
 */
template<typename dataT1, typename dataT2>
hipError_t elementwiseXxY_impl( dataT1 * x,
                                 dataT2 * y,
                                 int size
                               )
{

   hipError_t rv = hipSuccess;

   #ifdef __HIPCC__
   rv = hipGetLastError();
   elwiseMul<dataT1,dataT2><<<(size+255)/256, 256>>>( x, y, size);
   rv = hipGetLastError();
   #endif

   return rv;
}

template<>
hipError_t elementwiseXxY<float,float>( float * x,
                                         float * y,
                                         int size
                                       )
{
   return elementwiseXxY_impl<float,float>(x,y,size);
}

template<>
hipError_t elementwiseXxY<double,double>( double * x,
                                           double * y,
                                           int size
                                         )
{
   return elementwiseXxY_impl<double,double>(x,y,size);
}

template<>
hipError_t elementwiseXxY<hipComplex,float>( hipComplex * x,
                                             float * y,
                                             int size
                                           )
{
   return elementwiseXxY_impl<hipComplex,float>(x,y,size);
}

template<>
hipError_t elementwiseXxY<hipComplex,hipComplex>( hipComplex * x,
                                                 hipComplex * y,
                                                 int size
                                               )
{
   return elementwiseXxY_impl<hipComplex,hipComplex>(x,y,size);
}

template<>
hipError_t elementwiseXxY<hipDoubleComplex,double>( hipDoubleComplex * x,
                                                    double * y,
                                                    int size
                                                  )
{
   return elementwiseXxY_impl<hipDoubleComplex,double>(x,y,size);
}

template<>
hipError_t elementwiseXxY<hipDoubleComplex,hipDoubleComplex>( hipDoubleComplex * x,
                                                             hipDoubleComplex * y,
                                                             int size
                                                           )
{
   return elementwiseXxY_impl<hipDoubleComplex,hipDoubleComplex>(x,y,size);
}

//----------------------------------------------------
// Tgemv

template<>
hipblasStatus_t cublasTgemv<float>( hipblasHandle_t handle,   
                                   hipblasOperation_t trans, 
                                   int m,                   
                                   int n,                   
                                   const float *alpha,     
                                   const float *A,          
                                   int lda,                 
                                   const float *x,         
                                   int incx,                
                                   const float *beta,      
                                   float *y,                
                                   int incy                 
                                 )
{
   return ::hipblasSgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
}

template<>
hipblasStatus_t cublasTgemv<double>( hipblasHandle_t handle,   
                                    hipblasOperation_t trans, 
                                    int m,                   
                                    int n,                   
                                    const double *alpha,     
                                    const double *A,          
                                    int lda,                 
                                    const double *x,         
                                    int incx,                
                                    const double *beta,      
                                    double *y,                
                                    int incy                 
                                  )
{
   return ::hipblasDgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
}

template<>
hipblasStatus_t cublasTgemv<float>( hipblasHandle_t handle,   
                                   hipblasOperation_t trans, 
                                   int m,                   
                                   int n,                   
                                   const float *alpha,     
                                   const float *A,          
                                   const float *x,         
                                   const float *beta,      
                                   float *y                
                                 )
{
   return ::hipblasSgemv(handle, trans, m, n, alpha, A, m, x, 1, beta, y, 1);
}

template<>
hipblasStatus_t cublasTgemv<double>( hipblasHandle_t handle,   
                                    hipblasOperation_t trans, 
                                    int m,                   
                                    int n,                   
                                    const double *alpha,     
                                    const double *A,          
                                    const double *x, 
                                    const double *beta,      
                                    double *y                
                                  )
{
   return ::hipblasDgemv(handle, trans, m, n, alpha, A, m, x, 1, beta, y, 1);
}

}//namespace cuda 
}//namespace mx
